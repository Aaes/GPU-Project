#include<stdio.h>
#define _USE_MATH_DEFINES
#include<math.h>
#include<float.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include <hip/hip_runtime_api.h>
#include <time.h>

typedef struct {
	double x;
	double y;
} point;

//CPU vars
//stepsizes
double h1; //Stepsize on level 1 (steps per year)
double h2; //Stepsize on level 2 (steps per year)
double h3; //Stepsize on level 3 (steps per year)
double g; //Years form the insurance holders death to when the payment starts
double r; //retirement age
double x; //Age at the time of calculation (t = 0)
double level2fullsteps; //since the fullsteps in the middle model is constant it can be calculated in the main() method

#define imin(a,b) (a<b?a:b)
int threadsPerBlock;
int blocksPerGrid;

bool shouldPrint = 0;

#define N 7
//x,r,g,h1,h2,h3,h2fullstepsizes
__constant__ double constData[N];

point Outer();
point OuterRK(double x, double y, double h, double *temp, int index);
double OuterDiff(double px, double py, double middley);
__global__ void Middle(double *outerx, double *temp);
__device__ point Inner(double eta, double t, double k);
__device__ point InnerRK(double h, point p, double eta, double t, double k, double x);
__device__ double InnerDiff(double t, double s, double fs, double k, double eta);
__device__ double MiddleDiff(double tau, double eat, double InnerY);
__device__ double r_d(double t);
__host__ double r_(double t);
__device__ double k(double tau);
__device__ double gTau(double tau);
__host__ double GmFemale(double t);
__device__ double GmMale(double t);
__device__ double Parabel(double t);
__device__ double f(double eta, double tau);
double roundWithString(double n);

int main(int argc, char *argv[])
{
	hipSetDevice(0);
	clock_t start = clock();
	//CPU vars
	//stepsizes
	g = strtod(argv[1], NULL);
	r = strtod(argv[2], NULL);
	x = strtod(argv[3], NULL);
	h1 = strtod(argv[4], NULL);
	h2 = 2.0;
	h3 = strtod(argv[4], NULL);
	// h1 = 10.0; //Stepsize on level 1 (steps per year)
	// h2 = 2.0; //Stepsize on level 2 (steps per year)
	// h3 = 10.0; //Stepsize on level 3 (steps per year)
	// g = 10.0; //Years form the insurance holders death to when the payment starts
	// r = 65.0; //retirement age
	// x = 35.0; //Age at the time of calculation (t = 0)
	level2fullsteps = floor(119 * h2); //since the fullsteps in the middle model is constant it can be calculated in the main() method
	double h_data[N] = {x,r,g,h1,h2,h3,level2fullsteps};
	
	//Copy data to constData
	hipMemcpyToSymbol(HIP_SYMBOL(constData), h_data, N*sizeof(double));
	
	//Start solver
	point result = Outer();

	clock_t end = clock();
	double elapsedTime = (end-start)/(double)CLOCKS_PER_SEC;
	printf("%0.2f\n", elapsedTime);
	//printf("OuterPar\n");
	//printf("Execution time: %0.2f\n", elapsedTime);

	//print GPU result
	//printf("Result point: %.14f , %.14f \n" , result.x, result.y);
	//Prevent the cmd window from instantly closing
	//getc(stdin);

	//terminate
	return 0;
}

//the outer model
//Change outer.
//It needs to take a pointer to r, g and x and nextPoint, and change return type to void
point Outer(){
	
	double stepsize = -1 / h1; //since we are taking steps back the stepsize is negative
	
	int fullsteps = floor((120 - x) * h1); //the full amount of steps we need to take in this model
	
	//double firstStep = -(fullsteps * stepsize) - (120-x); //since the fullsteps is an int we need to take the remainder as a step first (if one exists)	
	int size = (fullsteps*2);

	//CPU vars:
	double *steps = new double[size];
	double *temp = new double[size];

	//GPU vars:
	double *d_steps;
	double *d_temp;

	//Prepare an array of all possible x values
	int stp = 0;
	int b; //fullstep counter
	for(b = 0; b <= size; b++)
	{
		//Take full step
		if(b%2 == 0){
			steps[b] = (120-x) + stp*(-1.0/h1);
		}
		//Take half step (value needed in OuterRK)
		else{
			steps[b] = ((120-x) + stp*(-1.0/h1))+((-1.0/h1)/2);
			stp += 1;
		}
	}

	//Set blocks pr grid and threads pr block
	blocksPerGrid = size;
	threadsPerBlock = 128;

	//printf("Blocks: %d\n", blocksPerGrid);
	//printf("Threads pr block: %d\n", threadsPerBlock);
	//printf("Ineffective threads in each block: %.1f\n", (level2fullsteps/threadsPerBlock)*threadsPerBlock-level2fullsteps);

	//Malloc and set memory on the GPU
	hipMalloc((void**)&d_steps, size*sizeof(double));
	hipMalloc((void**)&d_temp, size*sizeof(point));
	hipMemset(d_steps, 0.0, size*sizeof(double));
	hipMemset(d_temp, 0, size*sizeof(double));

	//Copy data to the GPU
	hipMemcpy(d_steps, steps, size*sizeof(double), hipMemcpyHostToDevice);

	//Start the kernel
	Middle<<<blocksPerGrid,threadsPerBlock, level2fullsteps*sizeof(double)>>>(d_steps, d_temp);
	//Check that everything went ok
	{
	    hipError_t cudaerr = hipDeviceSynchronize();
	    if (cudaerr != (hipError_t)hipSuccess)
	        printf("kernel launch failed with error \"%s\".\n",
	               hipGetErrorString(cudaerr));
	}
	//copy intermediate results back to CPU
	hipMemcpy(temp, d_temp, size*sizeof(double), hipMemcpyDeviceToHost);

	point nextPoint = {120.0-x,0.0}; //set the startpoint

	int s; //fullstep counter
	//Run outer as normal using intermediate results from the GPU
	for(s = 0; s <size; s+=2)
	{
		nextPoint = OuterRK(steps[s], nextPoint.y, stepsize, temp, s);
	}
	hipFree(d_temp);
	hipFree(d_steps);
	return nextPoint;
}

point OuterRK(double x, double y, double h, double *temp, int index){
 double k1 = h * OuterDiff(x, y, temp[index]);

 double k2 = h * OuterDiff(x + h/2.0, y + k1/2, temp[index+1]);

 double k3 = h * OuterDiff(x + h/2.0, y + k2/2, temp[index+1]);
 
 double k4 = h * OuterDiff(x + h, y + k3, temp[index+2]);
 
 double newy = y + k1/6 + k2/3 + k3/3 + k4/6;

 point resultPoint = {x + h,newy};
 return resultPoint;
}

//The differential equation for the outer model
double OuterDiff(double px, double py, double middley)
{
	return r_(px) * py - GmFemale(x + px) * (middley - py);
}

//the middle model
__global__ void Middle(double *outerx, double *temp){
	
	extern __shared__ double kSum[];

	int tid = threadIdx.x;
	double stepsize = -1.0 / constData[4]; //since we are taking steps back the stepsize is negative
	//since the fullsteps in the middle model is constant it can be calculated in the main() method
	const int fullsteps = constData[6];
	double gt = outerx[blockIdx.x];
	while(tid<fullsteps){
		double tau = constData[0] + gt;
		double kk = k(tau);
		double eta = (120.0+(tid*stepsize));
		double k1 = stepsize * MiddleDiff(tau, eta, Inner(eta, gt, kk).y);
		double k2 = stepsize * MiddleDiff(tau, eta + stepsize/2, Inner(eta + stepsize/2, gt, kk).y);		
		double k4 = stepsize * MiddleDiff(tau, eta + stepsize, Inner(eta + stepsize, gt, kk).y);
		kSum[tid] = k1/6+k2/3+k2/3+k4/6;
		//kSum[tid] = 0.0;
		tid = tid + blockDim.x;
	}
	tid = threadIdx.x;
	__syncthreads();

	if(tid == 0){
		double y = 0.0;
		int i;
		for(i = fullsteps-1; i>=0; i--){
			y = y+kSum[i];
		}
		temp[blockIdx.x] = y;
	}
}

//The differential equation for the middle model
__device__ double MiddleDiff(double tau, double eta, double innerY)
{
	return -1 * gTau(tau) * f(eta,tau) * innerY;
}

__device__ point Inner(double eta, double t, double k){
	
	double stepsize = -1.0 / constData[5]; //since we are taking steps back the stepsize is negative
	point nextPoint = {120-eta, 0}; //set the startpoint
	int fullsteps = floor((120 - eta) * constData[5]); //the full amount of steps we need to take in this model
	
	double firstStep = -(fullsteps * stepsize) - (120-eta); //since the fullsteps is an int we need to take the remainder as a step first (if one exists)
	nextPoint = InnerRK(firstStep, nextPoint, eta, t, k, (120-eta));

	//Solve the differential equation with the Runge-Kutta solver
	int s; //fullstep counter
	for(s = 0; s < fullsteps; s++)
	{	
		nextPoint = InnerRK(stepsize, nextPoint, eta, t, k, (((120.0-eta)+firstStep)+s*stepsize));
	}
	return nextPoint;
}

//the runge-kutta solver for the inner model
__device__ point InnerRK(double h, point p, double eta, double t, double k, double x){
	
	double k1 = h * InnerDiff(t, x, p.y, k, eta);
	
	double k2 = h * InnerDiff(t, (x + h/2), (p.y + k1/2), k, eta);		
	
	double k3 = h * InnerDiff(t, (x + h/2), (p.y + k2/2), k, eta);
	
	double k4 = h * InnerDiff(t, (x + h), (p.y + k3), k, eta);	
	
	double y = p.y + k1/6 + k2/3 + k3/3 + k4/6;
	
	point resultPoint = {x + h,y};
	return resultPoint;
}

//The differential equation for the inner model
__device__ double InnerDiff(double t, double s, double fs, double k, double eta)
{
	return r_d(t + s) * fs - (s >= k ? 1 : 0) - GmMale(eta + s) * (0 - fs);
}

//FUNCTIONS
//helper function to gTau. Taken from collectiveHelp.txt
__device__ double Parabel(double t){
	return fmax((15.0 - t) * (t - 120.0), 0.0);
}

//Function to determine g(tau)
__device__ double gTau(double tau)
{
	//taken from collectiveHelp.txt
	double marriageProbabilityPeak = 0.9;
	double scalefactor = Parabel(67.5) / marriageProbabilityPeak;
	
	return Parabel(tau) / scalefactor;
}

//Function to determine f(eta,tau)
__device__ double f(double eta, double tau){
	//taken from collectiveHelp.txt
	double sigma = 3.0;
    double preFactorInNormalDensity = 1.0 / (sqrt(2.0 * M_PI) * sigma);
    double factorInExponent = 1.0 / (2.0 * sigma * sigma);

	return 
      (eta <= 0.0 || eta >= 120.0) 
      ? 0.0
      : preFactorInNormalDensity * exp(-1.0 * (tau - eta) * (tau - eta) * factorInExponent);
}

//Function to determine k
__device__ double k(double tau) {
	double r = constData[1];
	double g = constData[2];
	if(tau < r) return g;
	if(r <= tau && tau < r + g) return (r + g - tau);
	if(r + g <= tau) return 0.0;
	return 0.0;
}

//R function - interest rate
__device__ double r_d(double t) {
	return 0.05;
}

//R function - interest rate
__host__ double r_(double t) {
	return 0.05;
}

// Gompertz-Makeham mortality intensities for Danish women
__host__ double GmFemale(double t) {
    return 0.0005 + pow(10.0, 5.728 - 10 + 0.038*(t));
}

// Gompertz-Makeham mortality intensities for Danish men
__device__ double GmMale(double t) {
    return 0.0005 + pow(10.0, 5.880 - 10 + 0.038*(t));
}

double roundWithString(double n)
{
char buffer [50];
sprintf (buffer, "%.13f", n);
return atof(buffer);
}
