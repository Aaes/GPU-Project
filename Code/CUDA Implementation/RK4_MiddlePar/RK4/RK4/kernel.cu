#include<stdio.h>
#define _USE_MATH_DEFINES
#include<math.h>
#include<float.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include <hip/hip_runtime_api.h>
#include <time.h>

typedef struct {
	double x;
	double y;
} point;

//CPU vars
//stepsizes
double h1; //Stepsize on level 1 (steps per year)
double h2; //Stepsize on level 2 (steps per year)
double h3; //Stepsize on level 3 (steps per year)
double g; //Years form the insurance holders death to when the payment starts
double r; //retirement age
double x; //Age at the time of calculation (t = 0)
double level2fullsteps; //since the fullsteps in the middle model is constant it can be calculated in the main() method

#define imin(a,b) (a<b?a:b)
const int threadsPerBlock = 32;
int blocksPerGrid;

#define N 7
//x,r,g,h1,h2,h3,h2fullstepsizes
__constant__ double constData[N];

point Outer();
point OuterRK(double h, point p);
double OuterDiff(double px, double py);
__global__ void Middle(double *t, double *kSum);
__device__ point Inner(double eta, double t, double k);
__device__ point InnerRK(double h, point p, double eta, double t, double k);
__device__ double InnerDiff(double t, double s, double fs, double k, double eta);
__device__ double MiddleDiff(double tau, double eat, double InnerY);
__device__ double r_d(double t);
__host__ double r_(double t);
__device__ double k(double tau);
__device__ double gTau(double tau);
__host__ double GmFemale(double t);
__device__ double GmMale(double t);
__device__ double Parabel(double t);
__device__ double f(double eta, double tau);

int main()
{
	clock_t start = clock();

	//CPU vars
	//stepsizes
	h1 = 10.0; //Stepsize on level 1 (steps per year)
	h2 = 2.0; //Stepsize on level 2 (steps per year)
	h3 = 10.0; //Stepsize on level 3 (steps per year)
	g = 31.0; //Years form the insurance holders death to when the payment starts
	r = 79.0; //retirement age
	x = 110.0; //Age at the time of calculation (t = 0)
	level2fullsteps = floor(120 * h2) - h2; //since the fullsteps in the middle model is constant it can be calculated in the main() method
	double h_data[N] = {x,r,g,h1,h2,h3,level2fullsteps};
	
	blocksPerGrid = level2fullsteps/threadsPerBlock + 1;
	printf("Blocks: %d\n", blocksPerGrid);
	printf("Threads pr block: %d\n", threadsPerBlock);
	printf("Ineffective threads in last block: %.1f\n", blocksPerGrid*threadsPerBlock-level2fullsteps);
	//Copy data to constData
	hipMemcpyToSymbol(HIP_SYMBOL(constData), h_data, N*sizeof(double));
	
	//Start solver
	point result = Outer();

	clock_t end = clock();
	double elapsedTime = (end-start)/(double)CLOCKS_PER_SEC;

	printf("Execution time: %0.2f\n", elapsedTime);

	//print GPU result
	printf("Result point: %.14f , %.14f \n" , result.x, result.y);
	//Prevent the cmd window from instantly closing
	getc(stdin);

	//terminate
	return 0;
}

//the outer model
//Change outer.
//It needs to take a pointer to r, g and x and nextPoint, and change return type to void
point Outer(){
	
	double stepsize = -1 / h1; //since we are taking steps back the stepsize is negative
	point nextPoint = {120.0-x,0.0}; //set the startpoint
	
	int fullsteps = floor((120 - x) * h1); //the full amount of steps we need to take in this model
	
	double firstStep = -(fullsteps * stepsize) - (120-x); //since the fullsteps is an int we need to take the remainder as a step first (if one exists)	
	nextPoint = OuterRK(firstStep, nextPoint);

	//Solve the differential equation with the Runge-Kutta solver
	int s; //fullstep counter
	for(s = 0; s < fullsteps; s++)
	{
		nextPoint = OuterRK(stepsize, nextPoint);
	}
	
	//printf("Nextpoint: %.14f , %.14f \n" , nextPoint.x, nextPoint.y);
	
	return nextPoint;
}

//the runge-kutta solver for the outer model
point OuterRK(double h, point p){
	
	double k1 = h * OuterDiff(p.x, p.y);
	
	double k2 = h * OuterDiff(p.x + h/2, p.y + k1/2);		
	
	double k3 = h * OuterDiff(p.x + h/2, p.y + k2/2);		

	double k4 = h * OuterDiff(p.x + h, p.y + k3);

	double y = p.y + k1/6 + k2/3 + k3/3 + k4/6;
	//printf("k = %f, %f, %f, %f\n", k1, k2, k3, k4);
	
	
	point resultPoint = {p.x + h,y};
	return resultPoint;
}

//The differential equation for the outer model
double OuterDiff(double px, double py)
{
	point p = {0.0, 0.0};
	point *result = &p;
	double *kSum = (double*)malloc(level2fullsteps * sizeof(double));


	//GPU vars
	double *d_px;
	double *d_kSum;

	//allocate __device__ memory for variables
	hipMalloc( (void**)&d_px, sizeof(double));
	hipMalloc( (void**)&d_kSum, level2fullsteps*sizeof(double));
	hipMemset	(d_kSum, 0, level2fullsteps*sizeof(double));

	hipMemcpy(d_px, &px, sizeof(double), hipMemcpyHostToDevice);
	//Start kernel
	Middle<<<blocksPerGrid,threadsPerBlock>>>(d_px, d_kSum);
	//Check that everything went ok
	{
	    hipError_t cudaerr = hipDeviceSynchronize();
	    if (cudaerr != hipSuccess)
	        printf("kernel launch failed with error \"%s\".\n",
	               hipGetErrorString(cudaerr));
	}

	//Copy the result of running the kernel back to the CPU
	hipMemcpy(kSum, d_kSum, level2fullsteps*sizeof(double), hipMemcpyDeviceToHost);

	point nextPoint = {120.0,0.0};
	double y = nextPoint.y;
	int i;
	for(i = level2fullsteps-1; i>=0; i--){
		y = y+kSum[i];
	}
	point res = {1.0, y};
	*result = res;

	hipFree(d_kSum);
	hipFree(d_px);
	free(kSum);

	return r_(px) * py - GmFemale(x + px) * ((*result).y - py);
}

//the middle model
__global__ void Middle(double *t, double *kSum){
	int tid = threadIdx.x + blockIdx.x*blockDim.x; //compute thread id here. Remove comment when done properly
	//since we are taking steps back the stepsize is negative
	double stepsize = -1 / constData[4]; //constData[4] refers to the h2 constant
	//since the fullsteps in the middle model is constant it can be calculated in the main() method
	const int fullsteps = constData[6]; //constData[6] refers to the level2fullsteps constant
	if(tid<fullsteps){
		double gt = *t; //de-reference input
		double tau = constData[0] + gt; //constData[0] refers to the x constant
		double kk = k(tau);
		double eta = (120+(tid*stepsize));
		double k1 = stepsize * MiddleDiff(tau, eta, Inner(eta, gt, kk).y);
		double k2 = stepsize * MiddleDiff(tau, eta + stepsize/2, Inner(eta + stepsize/2, gt, kk).y);		
		double k4 = stepsize * MiddleDiff(tau, eta + stepsize, Inner(eta + stepsize, gt, kk).y);
		kSum[tid] = k1/6+k2/3+k2/3+k4/6;
	}
}

//The differential equation for the middle model
__device__ double MiddleDiff(double tau, double eta, double innerY)
{
	return -1 * gTau(tau) * f(eta,tau) * innerY;
}

//the inner model
__device__ point Inner(double eta, double t, double k){
	
	double stepsize = -1 / constData[5]; //since we are taking steps back the stepsize is negative
	point nextPoint = {120-eta, 0}; //set the startpoint
	
	int fullsteps = floor((120 - eta) * constData[5]); //the full amount of steps we need to take in this model

	double firstStep = -(fullsteps * stepsize) - (120-eta); //since the fullsteps is an int we need to take the remainder as a step first (if one exists)
		
	nextPoint = InnerRK(firstStep, nextPoint, eta, t, k);

	//Solve the differential equation with the Runge-Kutta solver
	int s; //fullstep counter
	for(s = 0; s < fullsteps; s++)
	{
		nextPoint = InnerRK(stepsize, nextPoint, eta, t, k);
	}

	return nextPoint;
}

//the runge-kutta solver for the inner model
__device__ point InnerRK(double h, point p, double eta, double t, double k){
	
	double k1 = h * InnerDiff(t, p.x, p.y, k, eta);
	
	double k2 = h * InnerDiff(t, (p.x + h/2), (p.y + k1/2), k, eta);	
	
	double k3 = h * InnerDiff(t, (p.x + h/2), (p.y + k2/2), k, eta);
	
	double k4 = h * InnerDiff(t, (p.x + h), (p.y + k3), k, eta);
	
	double y = p.y + k1/6 + k2/3 + k3/3 + k4/6;
	
	point resultPoint = {p.x + h,y};
	return resultPoint;
}

//The differential equation for the inner model
__device__ double InnerDiff(double t, double s, double fs, double k, double eta)
{
	return r_d(t + s) * fs - (s >= k ? 1 : 0) - GmMale(eta + s) * (0 - fs);
}

//FUNCTIONS
//helper function to gTau. Taken from collectiveHelp.txt
__device__ double Parabel(double t){
	return fmax((15.0 - t) * (t - 120.0), 0.0);
}

//Function to determine g(tau)
__device__ double gTau(double tau)
{
	//taken from collectiveHelp.txt
	double marriageProbabilityPeak = 0.9;
	double scalefactor = Parabel(67.5) / marriageProbabilityPeak;
	
	return Parabel(tau) / scalefactor;
}

//Function to determine f(eta,tau)
__device__ double f(double eta, double tau){
	//taken from collectiveHelp.txt
	double sigma = 3.0;
    double preFactorInNormalDensity = 1.0 / (sqrt(2.0 * M_PI) * sigma);
    double factorInExponent = 1.0 / (2.0 * sigma * sigma);

	return 
      (eta <= 0.0 || eta >= 120.0) 
      ? 0.0
      : preFactorInNormalDensity * exp(-1.0 * (tau - eta) * (tau - eta) * factorInExponent);
}

//Function to determine k
__device__ double k(double tau) {
	double r = constData[1];
	double g = constData[2];
	if(tau < r) return g;
	if(r <= tau && tau < r + g) return (r + g - tau);
	if(r + g <= tau) return 0.0;
	return 0.0;
}

//R function - interest rate
__device__ double r_d(double t) {
	return 0.05;
}

//R function - interest rate
__host__ double r_(double t) {
	return 0.05;
}

// Gompertz-Makeham mortality intensities for Danish women
__host__ double GmFemale(double t) {
    return 0.0005 + pow(10, 5.728 - 10 + 0.038*(t));
}

// Gompertz-Makeham mortality intensities for Danish men
__device__ double GmMale(double t) {
    return 0.0005 + pow(10, 5.880 - 10 + 0.038*(t));
}
