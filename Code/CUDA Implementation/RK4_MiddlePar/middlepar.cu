#include<stdio.h>
#define _USE_MATH_DEFINES
#include<math.h>
#include<float.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include <hip/hip_runtime_api.h>
#include <time.h>

typedef struct {
	double x;
	double y;
} point;

//CPU vars
//stepsizes
double h1; //Stepsize on level 1 (steps per year)
double h2; //Stepsize on level 2 (steps per year)
double h3; //Stepsize on level 3 (steps per year)
double g; //Years form the insurance holders death to when the payment starts
double r; //retirement age
double x; //Age at the time of calculation (t = 0)
double level2fullsteps; //since the fullsteps in the middle model is constant it can be calculated in the main() method

#define imin(a,b) (a<b?a:b)
const int threadsPerBlock = 128;
int blocksPerGrid;

bool shouldPrint = 0;

#define N 7
//x,r,g,h1,h2,h3,h2fullstepsizes
__constant__ double constData[N];

point Outer();
point OuterRK(double x, double y, double h);
double OuterDiff(double px, double py);
__global__ void Middle(double *t, double *kSum);
__device__ point Inner(double eta, double t, double k);
__device__ point InnerRK(double h, point p, double eta, double t, double k, double x);
__device__ double InnerDiff(double t, double s, double fs, double k, double eta);
__device__ double MiddleDiff(double tau, double eat, double InnerY);
__device__ double r_d(double t);
__host__ double r_(double t);
__device__ double k(double tau);
__device__ double gTau(double tau);
__host__ double GmFemale(double t);
__device__ double GmMale(double t);
__device__ double Parabel(double t);
__device__ double f(double eta, double tau);

int main(int argc, char *argv[])
{
	hipSetDevice(0);
	clock_t start = clock();
	g = strtod(argv[1], NULL);
	r = strtod(argv[2], NULL);
	x = strtod(argv[3], NULL);
	h1 = strtod(argv[4], NULL);
	h2 = 2.0;
	h3 = strtod(argv[4], NULL);
	
	//CPU vars
	//stepsizes
	// h1 = 10.0; //Stepsize on level 1 (steps per year)
	// h2 = 2.0; //Stepsize on level 2 (steps per year)
	// h3 = 10.0; //Stepsize on level 3 (steps per year)
	// g = 10.0; //Years form the insurance holders death to when the payment starts
	// r = 65.0; //retirement age
	// x = 35.0; //Age at the time of calculation (t = 0)
	
	level2fullsteps = floor(119 * h2); //since the fullsteps in the middle model is constant it can be calculated in the main() method
	double h_data[N] = {x,r,g,h1,h2,h3,level2fullsteps};

	blocksPerGrid = level2fullsteps/threadsPerBlock + 1;
	//printf("Blocks: %d\n", blocksPerGrid);
	//printf("Threads pr block: %d\n", threadsPerBlock);
	//Copy data to constData
	hipMemcpyToSymbol(HIP_SYMBOL(constData), h_data, N*sizeof(double));

	//Start solver
	point result = Outer();

	clock_t end = clock();
	double elapsedTime = (end-start)/(double)CLOCKS_PER_SEC;
	printf("%0.2f\n", elapsedTime);
	//printf("MiddlePar\n");
	//printf("Execution time: %0.2f\n", elapsedTime);

	//print GPU result
	//printf("Result point: %.14f , %.14f \n" , result.x, result.y);
	//Prevent the cmd window from instantly closing
	//getc(stdin);

	//terminate
	return 0;
}

//It needs to take a pointer to r, g and x and nextPoint, and change return type to void
point Outer(){
	
	double stepsize = -1.0 / h1; //since we are taking steps back the stepsize is negative
	point nextPoint = {120.0-x,0.0}; //set the startpoint
	
	int fullsteps = floor((120 - x) * h1); //the full amount of steps we need to take in this model

	//Solve the differential equation with the Runge-Kutta solver
	int s; //fullstep counter
	for(s = 0; s < fullsteps; s++)
	{
		nextPoint = OuterRK((120-x) - s*(1.0/h1), nextPoint.y, stepsize);
	}
	
	return nextPoint;
}

//the runge-kutta solver for the outer model
point OuterRK(double x, double y, double h){
 
 double k1 = h * OuterDiff(x, y);

 double k2 = h * OuterDiff(x + h/2.0, y + k1/2);

 double k3 = h * OuterDiff(x + h/2.0, y + k2/2);
 
 double k4 = h * OuterDiff(x + h, y + k3);

 double newy = y + k1/6 + k2/3 + k3/3 + k4/6;

 point resultPoint = {x + h,newy};
 return resultPoint;
}

//The differential equation for the outer model
double OuterDiff(double px, double py)
{
	double *kSum = (double*)malloc(level2fullsteps * sizeof(double));

	//GPU vars
	double *d_px;
	double *d_kSum;

	//allocate __device__ memory for variables
	hipMalloc( (void**)&d_px, sizeof(double));
	hipMalloc( (void**)&d_kSum, level2fullsteps*sizeof(double));
	hipMemset	(d_kSum, 0, level2fullsteps*sizeof(double));

	hipMemcpy(d_px, &px, sizeof(double), hipMemcpyHostToDevice);
	//Start kernel
	Middle<<<blocksPerGrid,threadsPerBlock>>>(d_px, d_kSum);
	//Check that everything went ok
	{
	    hipError_t cudaerr = hipDeviceSynchronize();
	    if (cudaerr != (hipError_t)hipSuccess)
	        printf("kernel launch failed with error \"%s\".\n",
	               hipGetErrorString(cudaerr));
	}

	//Copy the result of running the kernel back to the CPU
	hipMemcpy(kSum, d_kSum, level2fullsteps*sizeof(double), hipMemcpyDeviceToHost);

	point nextPoint = {120.0,0.0};
	double y = nextPoint.y;
	int i;
	for(i = level2fullsteps-1; i>=0; i--){
		y = y+kSum[i];
	}

	hipFree(d_kSum);
	hipFree(d_px);
	free(kSum);

	return r_(px) * py - GmFemale(x + px) * (y - py);
}

//the middle model
__global__ void Middle(double *t, double *kSum){

	int tid = threadIdx.x + blockIdx.x*blockDim.x; //compute thread id here. Remove comment when done properly
	//since we are taking steps back the stepsize is negative
	double stepsize = -1 / constData[4]; //constData[4] denotes h2
	//since the fullsteps in the middle model is constant it can be calculated in the main() method
	const int fullsteps = constData[6];
	if(tid<fullsteps){
		double gt = *t;
		double tau = constData[0] + gt;
		double kk = k(tau);
		double eta = (120.0+(tid*stepsize));
		double k1 = stepsize * MiddleDiff(tau, eta, Inner(eta, gt, kk).y);
		double k2 = stepsize * MiddleDiff(tau, eta + stepsize/2, Inner(eta + stepsize/2, gt, kk).y);		
		double k4 = stepsize * MiddleDiff(tau, eta + stepsize, Inner(eta + stepsize, gt, kk).y);
		kSum[tid] = k1/6+k2/3+k2/3+k4/6;
	}
}

//The differential equation for the middle model
__device__ double MiddleDiff(double tau, double eta, double innerY)
{
	return -1 * gTau(tau) * f(eta,tau) * innerY;
}

//the inner model
__device__ point Inner(double eta, double t, double k){
	
	double stepsize = -1.0 / constData[5]; //since we are taking steps back the stepsize is negative
	point nextPoint = {120-eta, 0}; //set the startpoint
	int fullsteps = floor((120 - eta) * constData[5]); //the full amount of steps we need to take in this model
	
	double firstStep = -(fullsteps * stepsize) - (120-eta); //since the fullsteps is an int we need to take the remainder as a step first (if one exists)
	nextPoint = InnerRK(firstStep, nextPoint, eta, t, k, (120-eta));

	//Solve the differential equation with the Runge-Kutta solver
	int s; //fullstep counter
	for(s = 0; s < fullsteps; s++)
	{
		if(nextPoint.x > 70){
		}
		nextPoint = InnerRK(stepsize, nextPoint, eta, t, k, (((120.0-eta)+firstStep)+s*stepsize));
	}
	return nextPoint;
}

//the runge-kutta solver for the inner model
__device__ point InnerRK(double h, point p, double eta, double t, double k, double x){
	
	double k1 = h * InnerDiff(t, x, p.y, k, eta);
	
	double k2 = h * InnerDiff(t, (x + h/2), (p.y + k1/2), k, eta);		
	
	double k3 = h * InnerDiff(t, (x + h/2), (p.y + k2/2), k, eta);
	
	double k4 = h * InnerDiff(t, (x + h), (p.y + k3), k, eta);	
	
	double y = p.y + k1/6 + k2/3 + k3/3 + k4/6;
	
	point resultPoint = {x + h,y};
	return resultPoint;
}

//The differential equation for the inner model
__device__ double InnerDiff(double t, double s, double fs, double k, double eta)
{
	return r_d(t + s) * fs - (s >= k ? 1 : 0) - GmMale(eta + s) * (0 - fs);
}

//FUNCTIONS
//helper function to gTau. Taken from collectiveHelp.txt
__device__ double Parabel(double t){
	return fmax((15.0 - t) * (t - 120.0), 0.0);
}

//Function to determine g(tau)
__device__ double gTau(double tau)
{
	//taken from collectiveHelp.txt
	double marriageProbabilityPeak = 0.9;
	double scalefactor = Parabel(67.5) / marriageProbabilityPeak;

	return Parabel(tau) / scalefactor;
}

//Function to determine f(eta,tau)
__device__ double f(double eta, double tau){
	//taken from collectiveHelp.txt
	double sigma = 3.0;
    double preFactorInNormalDensity = 1.0 / (sqrt(2.0 * M_PI) * sigma);
    double factorInExponent = 1.0 / (2.0 * sigma * sigma);

	return 
      (eta <= 0.0 || eta >= 120.0) 
      ? 0.0
      : preFactorInNormalDensity * exp(-1.0 * (tau - eta) * (tau - eta) * factorInExponent);
}

//Function to determine k
__device__ double k(double tau) {
	double r = constData[1];
	double g = constData[2];
	if(tau < r) return g;
	if(r <= tau && tau < r + g) return (r + g - tau);
	if(r + g <= tau) return 0.0;
	return 0.0;
}

//R function - interest rate
__device__ double r_d(double t) {
	return 0.05;
}

//R function - interest rate
__host__ double r_(double t) {
	return 0.05;
}

// Gompertz-Makeham mortality intensities for Danish women
__host__ double GmFemale(double t) {
    return 0.0005 + pow(10.0, 5.728 - 10 + 0.038*(t));
}

// Gompertz-Makeham mortality intensities for Danish men
__device__ double GmMale(double t) {
    return 0.0005 + pow(10.0, 5.880 - 10 + 0.038*(t));
}
